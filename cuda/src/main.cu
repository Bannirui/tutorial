#include <iostream>
#include <hip/hip_runtime.h>

__global__ void AddIntsCUDA(int *a, int *b) {
    a[0]+=b[0];
}

int main() {
    int a=5, b=8;
    int *p_a, *p_b;
    hipMalloc(&p_a, sizeof(int));
    hipMalloc(&p_b, sizeof(int));

    hipMemcpy(p_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(p_b, &b, sizeof(int), hipMemcpyHostToDevice);

    AddIntsCUDA<<<1, 1>>>(p_a, p_b);

    hipMemcpy(&a, p_a, sizeof(int), hipMemcpyDeviceToHost);

    std::cout<<"the result is "<<a<<std::endl;

    hipFree(p_a);
    hipFree(p_b);
    return 0;
}
