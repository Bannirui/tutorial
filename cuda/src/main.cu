#include <iostream>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>

/**
 * kernel called by host, executed in device
 * add 2 nums
 */
__global__ void AddInt(int *a, int *b) {
    for (int i = 0; i < 10000005; i++) {
        a[0] += b[0];
    }
}

__global__ void AddInts(int *a, int *b, int count) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < count) {
        a[id] += b[id];
    }
}

void test2NUms() {
    int a = 0, b = 1;
    int *p_a, *p_b;
    if (hipMalloc(&p_a, sizeof(int)) != hipSuccess) {
        std::cout << "error allocating memory" << std::endl;
        return;
    }
    if (hipMalloc(&p_b, sizeof(int)) != hipSuccess) {
        std::cout << "error allocating memory" << std::endl;
        return;
    }

    if (hipMemcpy(p_a, &a, sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
        std::cout << "error cpy memory" << std::endl;
        hipFree(p_a);
        hipFree(p_b);
        return;
    }
    if (hipMemcpy(p_b, &b, sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
        std::cout << "error cpy memory" << std::endl;
        hipFree(p_a);
        hipFree(p_b);
        return;
    }

    AddInt<<<1, 1>>>(p_a, p_b);

    hipMemcpy(&a, p_a, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "the result is " << a << std::endl;

    hipFree(p_a);
    hipFree(p_b);
}

void testThreads() {
    srand(time(nullptr));
    int count = 100;
    int *h_a = new int[count];
    int *h_b = new int[count];

    for (int i = 0; i < count; i++) {
        h_a[i] = rand() % 1000;
        h_b[i] = rand() % 1000;
    }
    std::cout << "Prior to addition:" << std::endl;
    for (int i = 0; i < 5; i++) {
        std::cout << h_a[i] << " " << h_b[i] << std::endl;
    }
    int *d_a, *d_b;
    if (hipMalloc(&d_a, sizeof(int) * count) != hipSuccess) {
        std::cout << "malloc error" << std::endl;
        return;
    }
    if (hipMalloc(&d_b, sizeof(int) * count) != hipSuccess) {
        std::cout << "malloc error" << std::endl;
        hipFree(d_a);
        return;
    }
    if (hipMemcpy(d_a, h_a, sizeof(int) * count, hipMemcpyHostToDevice) != hipSuccess) {
        std::cout << "hipMemcpy error" << std::endl;
        hipFree(d_a);
        hipFree(d_b);
        return;
    }
    if (hipMemcpy(d_b, h_b, sizeof(int) * count, hipMemcpyHostToDevice) != hipSuccess) {
        std::cout << "hipMemcpy error" << std::endl;
        hipFree(d_a);
        hipFree(d_b);
        return;
    }
    AddInts<<<count / 256 + 1,256>>>(d_a, d_b, count);
    if (hipMemcpy(h_a, d_a, sizeof(int) * count, hipMemcpyDeviceToHost) != hipSuccess) {
        delete[] h_a;
        delete[] h_b;
        hipFree(d_a);
        hipFree(d_b);
        std::cout << "hipMemcpy error" << std::endl;
        return;
    }
    // first 5
    for (int i = 0; i < 5; i++) {
        std::cout << "It's " << h_a[i] << std::endl;
    }
    hipFree(d_a);
    hipFree(d_b);
    delete[] h_a;
    delete[] h_b;
}

int main() {
    test2NUms();
    testThreads();
    return 0;
}
