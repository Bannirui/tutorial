#include <iostream>
#include <hip/hip_runtime.h>

__global__ void AddIntsCUDA(int *a, int *b) {
    for (int i = 0; i < 10000005; i++) {
        a[0] += b[0];
    }
}

int main() {
    int a = 0, b = 1;
    int *p_a, *p_b;
    if (hipMalloc(&p_a, sizeof(int)) != hipSuccess) {
        std::cout << "error allocating memory" << std::endl;
        return 0;
    }
    if (hipMalloc(&p_b, sizeof(int)) != hipSuccess) {
        std::cout << "error allocating memory" << std::endl;
        return 0;
    }

    if (hipMemcpy(p_a, &a, sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
        std::cout << "error cpy memory" << std::endl;
        hipFree(p_a);
        hipFree(p_b);
        return 0;
    }
    if (hipMemcpy(p_b, &b, sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
        std::cout << "error cpy memory" << std::endl;
        hipFree(p_a);
        hipFree(p_b);
        return 0;
    }

    AddIntsCUDA<<<1, 1>>>(p_a, p_b);

    hipMemcpy(&a, p_a, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "the result is " << a << std::endl;

    hipFree(p_a);
    hipFree(p_b);
    return 0;
}
